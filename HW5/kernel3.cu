#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define group 25

int bx;
int by;

void test(int *img, int X, int Y) {

    FILE *fp;
    int i, j;

    fp = fopen("log.txt", "w");
    for(i=0;i<Y;i++){
	for(j=0;j<X;j++){
//	    printf("j:%d ", j);
	    fprintf(fp, "%d ", img[i*X+j]);
	}
//	printf("\n");
	fprintf(fp, "\n");
    }
    fclose(fp);

}

__global__ void mandelKernel(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, size_t pitch, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int bk;
    for(bk=0;bk<group;bk++){
	    
	    float x = lowerX + col * stepX;
	    float y = lowerY + row * stepY;

	    int i;
	    float z_re = x, z_im = y;
	    for (i = 0; i < maxIterations; ++i)
	    {

			if (z_re * z_re + z_im * z_im > 4.f)
				break;

			float new_re = z_re * z_re - z_im * z_im;
			float new_im = 2.f * z_re * z_im;
			z_re = x + new_re;
			z_im = y + new_im;
		}
		int* index = (int*)((char*)img+row*pitch);
		index[col] = i;

		row += blockDim.y*gridDim.y;
    }

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    //int bx = (upperX-lowerX+THREAD-1)/THREAD;
    //int by = (upperY-lowerY+THREAD-1)/THREAD;

    //dim3 threads(THREAD, THREAD);
    //dim3 blocks(bx, by);

    int *h_img;
    int *d_img;

//    int bytes = sizeof(float)*resX*resY;
    int bytes = resX*resY;

//    cudaMallocHost(h_img, bytes);
//    cudaMemcpy(h_img, img, bytes, cudaMemcpyHostToHost);
//    cudaMemcpyDeviceToHost(h_img, img, cudaMemcpyHostToHost);
  //  cudaMalloc(d_img, bytes);

    hipHostAlloc( (void**)&h_img, sizeof(int)*bytes ,hipHostMallocDefault);
    size_t pitch;
    hipMallocPitch((void**)&d_img, (size_t *)&pitch, sizeof(int)*resX, resY);

    printf("pitch: %lx\n", pitch);
//cudaMemcpy2D(   host_memory, /* dest */
//		  100*sizeof(float)   /*no pitch on host*/,
//                myArray,     /* src */
//		  pitch/*CUDA pitch*/,
//		  100*sizeof(float)/*width in bytes*/, 
//		  100/*heigth*/, 
//		  cudaMemcpyDeviceToHost);

//    cudaHostAlloc( (void**)&h_img, sizeof(float)*bytes ,cudaHostAllocDefault);
//    h_img = (float*)malloc(sizeof(float)*bytes);
    hipMemcpy2D(d_img, pitch, h_img, sizeof(int)*resX, sizeof(int)*resX, resY,hipMemcpyHostToDevice);

    printf("resX: %d, resY: %d\n", resX, resY);

    int THREAD = 16;
    bx = (resX)/THREAD;
    by = (resY)/THREAD;
    
	printf("bx: %d, by: %d\n", bx, by);
	by = (resY/group)/THREAD;
	printf("bx: %d, by: %d\n", bx, by);

	    	
    dim3 threads(THREAD, THREAD);
    dim3 blocks(bx, by) ;
    mandelKernel<<<blocks, threads>>>(upperX, upperY, lowerX, lowerY, d_img, resX, resY, pitch, maxIterations);
    //cudaMemcpy(img, d_img, bytes, cudaMemcpyDeviceToHost);
    hipMemcpy2D(h_img, sizeof(int)*resX, d_img, pitch, sizeof(int)*resX, resY,hipMemcpyDeviceToHost);

    hipMemcpy(img, h_img, sizeof(int)*bytes, hipMemcpyHostToHost);

//    cpy(img, h_img, resX, resY);
    test(h_img, resX, resY);
//    cudaFree(d_img);
    hipHostFree(h_img);
    //free(h_img);
}
